/*
* sequencealign_parallel.cu
*
* IMPORTANT: 
* 
* The final version of this code has been developed by Mustafa ACIKARAOĞLU, Mustafa SARAÇ,
* Mustafa Mert ÖGETÜRK as term project of Parallel Programming (COMP 429) course. 
* Koç University's code of ethics can be applied to this code and liability can not be 
* accepted for any negative situation. Therefore, be careful when you get content from here.
*
* This parallel version of Sequence Alignment code has been 
* implemented using the source specified in the link below. 
* 
* Reference:
* Implementation of Sequence Alignment in C++ 
* URL: <https://codereview.stackexchange.com/questions/97825/implementation-of-sequence-alignment-in-c>
* 
* NOTE:
*  
* THIS SOURCE CODE CONTAINS TWO VERSIONS OF
* PARALLELIZATION PROCESS.
* 
* FIRST VERSION: 
* ONLY TWO PART OF THE SERIAL IMPLEMENTATION IS PARALLELIZED
* IN A SUCCESSFUL WAY. THIS IMPLEMENTATION CONTAINS 
* 'alphabet_matching_penalty' AND 'array_filling_1' FUNCTIONS. 
* 
* SECOND VERSION:
* AS WE MENTIONED IN THE FINAL REPORT, WE TRIED TO IMPLEMENT
* THE PARALLELIZED VERSION OF THE PART OF THE 'align' FUNCTION
* FROM THE SERIAL IMPLEMENTATION, WHICH DIAGONALLY TRAVERSES 
* THROUGH THE END OF THE ARRAY BY FINDING THE MINIMUM OF THE 
* CURRENT INDEX'S LEFT, TOP AND LEFT-TOP INDEXES.
* HOWEVER, BECAUSE OF THE RACE CONDITION, WE TRIED TO FOLLOW
* MANY DIFFERENT WAYS TO SOLVE THIS ISSUE, BUT WE COULD NOT
* SUCCEED IT. THEREFORE, WE COMMENTED OUT ALL OF THE CODE THAT
* IS RELATED TO THE 'align_filling_2 kernel'.
*
*
* For more detailed questions you can review our project report.
*  
* You can also contact me at this email address: msarac13@ku.edu.tr
* 
*/ 

#include <iostream>
#include <string>
#include <vector>
#include <sys/time.h>
#include <stdio.h>
#include <stdlib.h>
#include <fstream>
#include <sstream>
#include <string.h>

#include <hip/hip_runtime.h>


using namespace std;

const size_t alphabets = 26;
static const double kMicro = 1.0e-6;

/*
 * Returns the current time
 */
double get_time();

/*
 * Loading a file into a char array
 */
char* load_file(char const* path);

/*
 * alpha_d[i][j] = penalty for matching the ith alphabet with the
 *               jth alphabet.
 * Here: Penalty for matching an alphabet with anoter one is 1
 *       Penalty for matching an alphabet with itself is 0
 */
__global__ void alphabet_matching_penalty(int *alpha_d);

/*
 * Returns the minimum integer
 */
int min(int a, int b, int c);

/*
 * Filling the first row and the first
 * column of the array based on the gap
 * penalty, which is equal to 2.
 */
__global__ void align_filling_1(size_t n, size_t m, int *A, int alpha_gap);

/*
 * COMMENTED OUT:
 *
 * Align_filling_2 is diagonally traversing by 
 * finding the minimum value among the current
 * index's left, top and left-top indexes      
 * through the end of the array. 

__global__ void align_filling_2(size_t n, size_t m, char* input_1_d,
    char* input_2_d, int *alpha_d, int *A, int alpha_gap);
*/

int main()
{
    double time_0, time_1, time_2, time_3, time_4, time_5;

    int *alpha_h, *alpha_d, *array_h, *array_d;
    char *input_1, *input_2;
    string a_aligned, b_aligned;

    /* 
     * COMMENTED OUT:
     *
     * Device char arrays that will be used in the 
     * align_filling_2 kernel.

    char *input_1_d, *input_2_d;

    */

    time_0 = get_time();

    // Reading the input strings that need to be aligned
    input_1 = load_file("DNA_Sequence_1.txt");
    input_2 = load_file("DNA_Sequence_2.txt");

    size_t n = strlen(input_1);
    size_t m = strlen(input_2);

    // Penalty for any alphabet matched with a gap
    int gap_penalty = 2;

    // Allocation
    alpha_h = (int *) malloc(sizeof(int) * alphabets * alphabets);
    array_h = (int *) malloc(sizeof(int) * (n + 1) * (m + 1));

    if(hipSuccess != hipMalloc((void**) &array_d, sizeof(int) * (n + 1) * (m + 1))){
        cout << "Cuda Malloc error for array_d." << endl;
    }

    if(hipSuccess != hipMalloc((void**) &alpha_d, sizeof(int) * alphabets * alphabets)){
        cout << "Cuda Malloc error for alpha_d." << endl;
    }

    /*
     * COMMENTED OUT:
     *
     * Memory Allocations for the arrays that will be used in the 
     * align_filling_2 kernel.
     *

    if(cudaSuccess != cudaMalloc((void**) &input_1_d, sizeof(int) * n)){
        cout << "Cuda Malloc error for input_1_d." << endl;
    }

    if(cudaSuccess != cudaMalloc((void**) &input_2_d, sizeof(int) * m)){
        cout << "Cuda Malloc error for input_2_d." << endl;
    }

    */


    // MEMORY COPYING FROM HOST TO THE DEVICE
    if(hipSuccess != hipMemcpy(array_d, array_h, sizeof(int) * (n + 1) * (m + 1), hipMemcpyHostToDevice)){
        cout << "Cuda Memory Copying error from array_h to array_d." << endl;
    }      

    if(hipSuccess != hipMemcpy(alpha_d, alpha_h, sizeof(int) * alphabets * alphabets, hipMemcpyHostToDevice)){
        cout << "Cuda Memory Copying error from alpha_h to alpha_d." << endl;
    }

    /*
     * COMMENTED OUT:
     *
     * Memory Copying from Host to Device for the arrays that will be used 
     * in the align_filling_2 kernel.
     *

    if(cudaSuccess != cudaMemcpy(input_1_d, input_1, sizeof(int) * n, cudaMemcpyHostToDevice)){
        cout << "Cuda Memory Copying error from input_1 to input_1_d." << endl;
    }

    if(cudaSuccess != cudaMemcpy(input_2_d, input_2, sizeof(int) * m, cudaMemcpyHostToDevice)){
        cout << "Cuda Memory Copying error from input_2 to input_2_d." << endl;
    }

    */
    
    time_1 = get_time();


    dim3 threads(alphabets, alphabets, 1);
    dim3 grid(alphabets * alphabets/threads.x, alphabets * alphabets/threads.y);

    alphabet_matching_penalty<<<grid, threads>>>(alpha_d);

    hipDeviceSynchronize();

    if(hipGetLastError() != hipSuccess){
        cout << "Kernel alphabet_matching_penalty was not launched." << endl;
    }

    time_2 = get_time();

    dim3 threads1(16, 16, 1);
    dim3 grid1(n / threads1.x, m / threads1.y);

    align_filling_1<<<grid1,threads1>>>(n, m, array_d, gap_penalty);

    hipDeviceSynchronize();

    if(hipGetLastError() != hipSuccess){
        cout << threads1.x << endl;
        cout << threads1.y << endl;
        cout << "Kernel filling_1 was not launched." << endl;
    }

    time_3 = get_time();

    /*
     * COMMENTED OUT:
     *
     * Memory Allocations for the arrays that will be used in the 
     * align_filling_2 kernel.
     *
    align_filling_2<<<grid1,threads1>>>(n, m, input_1_d, input_2_d, alpha_d, array_d, gap_penalty);

    cudaDeviceSynchronize();

    if(cudaGetLastError() != cudaSuccess){
        cout << threads1.x << endl;
        cout << threads1.y << endl;
        cout << "Kernel filling_2 was not launched." << endl;
    }

    time_4 = get_time();
    */

    // MEMORY COPYING FROM DEVICE TO THE HOST
    if(hipSuccess != hipMemcpy(array_h, array_d, sizeof(int) * (n + 1) * (m + 1), hipMemcpyDeviceToHost)){
        cout << "Cuda Memory Copying error from array_d to array_h." << endl;
    }

    if(hipSuccess != hipMemcpy(alpha_h, alpha_d, sizeof(int) * alphabets * alphabets, hipMemcpyDeviceToHost)){
        cout << "Cuda Memory Copying error from alpha_d to alpha_h." << endl;
    }

    /*
     * COMMENTED OUT:
     *
     * Memory Copying from Device to Host for the arrays, which were 
     * used in the align_filling_2 kernel, 
     *

    if(cudaSuccess != cudaMemcpy(input_1_d, input_1, sizeof(int) * n, cudaMemcpyDeviceToHost)){
        cout << "Cuda Memory Copying error from input_1 to input_1_d." << endl;
    }

    if(cudaSuccess != cudaMemcpy(input_2_d, input_2, sizeof(int) * m, cudaMemcpyDeviceToHost)){
        cout << "Cuda Memory Copying error from input_2 to input_2_d." << endl;
    }

    */


    /*
     * After the first filling step is finished,
     * the function is diagonally traversing by 
     * finding the minimum value among the current
     * index's left, top and left-top indexes 
     * through the end of the array. 
     */
    for (size_t i = 1; i <= n; ++i)
    {
        for (size_t j = 1; j <= m; ++j)
        {
            char x_i = input_1[i-1];
            char y_j = input_2[j-1];
            array_h[i * n + j] = min(array_h[(i-1) * n + (j-1)] + alpha_h[(x_i - 'A') * alphabets + (y_j - 'A')],
                          array_h[(i-1) * n + j] + gap_penalty,
                          array_h[i * n + (j-1)] + gap_penalty);
        }
    }


    /*
     * After the second filling step is finished,
     * the function is diagonally tracebacking 
     * through the beginning of the array and it 
     * is generating the output strings, which are
     * the aligned DNA sequences.
     */
    long k;

    size_t i = n;
    size_t j = m;

    for (; i >= 1 && j >= 1; --i)
    {
      k= i * n + j;

        char x_i = input_1[i-1];
        char y_j = input_2[j-1];
        if (array_h[k] == array_h[(i-1)*n + (j-1)] + alpha_h[(x_i - 'A') * alphabets + (y_j - 'A')])
        {
            a_aligned = x_i + a_aligned;
            b_aligned = y_j + b_aligned;
            --j;
        }
        else if (array_h[k] == array_h[(i-1)*n + j] + gap_penalty)
        {
            a_aligned = x_i + a_aligned;
            b_aligned = '-' + b_aligned;
        }
        else
        {
            a_aligned = '-' + a_aligned;
            b_aligned = y_j + b_aligned;
            --j;
        }
    }

    while (i >= 1 && j < 1)
    {
        a_aligned = input_1[i-1] + a_aligned;
        b_aligned = '-' + b_aligned;
        --i;
    }
    while (j >= 1 && i < 1)
    {
        a_aligned = '-' + a_aligned;
        b_aligned = input_2[j-1] + b_aligned;
        --j;
    }

    time_4 = get_time();

    /*
     * Needleman Score that represents the similarity 
     * between the DNA sequences.
     */
    int needleman_score = array_h[n * m - 1];

    ofstream outputFile;
    outputFile.open("output_file_cuda_v1.txt");
    outputFile << a_aligned << endl << b_aligned << endl;
    outputFile.close();
    
    free(alpha_h);
    free(array_h);

    hipFree(alpha_d);
    hipFree(array_d);

    /*
     * COMMENTED OUT:
     *
     * Freeing the device arrays

    cudaFree(input_1_d);
    cudaFree(input_2_d);
    */

    time_5 = get_time();

    // print
    printf("Time for mallocs and memcopies: %9.6f s\n", (time_1 - time_0));
    printf("Time for alphabet_matching_penalty: %9.6f s\n", (time_2 - time_1));
    printf("Time for filling_1: %9.6f s\n", (time_3 - time_2));
    printf("Time for filling_2 and get_traceback: %9.6f s\n", (time_4 - time_3));
    printf("Needleman score : %d\n",needleman_score);
    printf("Total time: %9.6f s\n", (time_5 - time_0));

    return 0;
}

double get_time() {
    struct timeval TV;
    struct timezone TZ;
    const int RC = gettimeofday(&TV, &TZ);
    if(RC == -1) {
        printf("ERROR: Bad call to gettimeofday\n");
        return(-1);
    }
    return( ((double)TV.tv_sec) + kMicro * ((double)TV.tv_usec) );
}

char* load_file(char const* path) {
    char* buffer = 0;
    long length;
    FILE * f = fopen (path, "rb"); //was "rb"

    if (f)
    {
      fseek (f, 0, SEEK_END);
      length = ftell (f);
      fseek (f, 0, SEEK_SET);
      buffer = (char*)malloc ((length+1)*sizeof(char));
      if (buffer)
      {
        fread (buffer, sizeof(char), length, f);
      }
      fclose (f);
    }
    buffer[length] = '\0';
    if(strlen(buffer) == 1){ printf("Failed to read the file"); }

    return buffer;
}

__global__ void alphabet_matching_penalty(int *alpha_d)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;
    long k;

    if(i < alphabets && j < alphabets){
        k = i * alphabets + j;
        if (i == j) {
            alpha_d[k] = 0;
        } else {
            alpha_d[k] = 1;
        }
     } else {
        return;
     }
}

int min(int a, int b, int c)
{
    return std::min(std::min(a,b), c);
}

__global__ void align_filling_1(size_t n, size_t m, int *A, int alpha_gap)
{
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(i <= n && j <= m){
        A[i*n] = alpha_gap * i;
        A[j] = alpha_gap * j;
    } else {
        return;
    }
}

/*
 * COMMENTED OUT:
 * 
 * Align_filling_2 is diagonally traversing by 
 * finding the minimum value among the current
 * index's left, top and left-top indexes      
 * through the end of the array. 

__global__ void align_filling_2(size_t n, size_t m, char* input_1_d,
    char* input_2_d, int *alpha_d, int *A, int alpha_gap)
{

    int i = blockIdx.x * blockDim.x + threadIdx.x;
    int j = blockIdx.y * blockDim.y + threadIdx.y;

    if(((i >=1) && (i <= n)) && ((j >= 1) && (j <= m))){
            char x_i = input_1_d[i-1];
            char y_j = input_2_d[j-1];

            int first = A[(i-1) * n + (j-1)] + alpha_d[(x_i - 'A') * alphabets + (y_j - 'A')];
            int second = A[(i-1) * n + j] + alpha_gap;
            int third = A[i * n + (j-1)] + alpha_gap;
         
            if(first < second && first < third){
                A[i * n + j] = first;
            } else if(second < first && second < third){
                A[i * n + j] = second;
            } else if(third < first && third < second){
                A[i * n + j] = third;
            }

        }

}

*/
